#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "Dotters.h"

#include <stdio.h>
#include <stdexcept>
#include <numeric>


#pragma region Error Handling macros
#define COOLDAE(x) { if (x != hipSuccess) throw std::runtime_error(#x); }
#define COOLDAEG(x) { hipError_t status; status = x; if (status != hipSuccess) throw std::runtime_error(hipGetErrorString(status)); }
#define COOLDAG(x, l) { if (x != hipSuccess) { fprintf(stderr, #x); goto l; }}
#pragma endregion

static int *deva = nullptr, *devb = nullptr, *devc = nullptr;

__global__ void mulKernel(int *productbuf, const int *arg1buf, const int *arg2buf)
{
	int i = threadIdx.x;
	productbuf[i] = arg1buf[i] * arg2buf[i];
}

typedef void* cudaBuf;

void cudaSetup(unsigned int size)
{
	COOLDAE(hipSetDevice(0));
	COOLDAE(hipMalloc((void**)&deva, size * sizeof(int)));
	COOLDAE(hipMalloc((void**)&devb, size * sizeof(int)));
	COOLDAE(hipMalloc((void**)&devc, size * sizeof(int)));
}

void cudaTransfer(cudaBuf cudabuf, const int *hostdata, unsigned int size)
{
	COOLDAE(hipMemcpy(cudabuf, hostdata, size * sizeof(int), hipMemcpyHostToDevice));
}

void cudaGetBack(int *hostdata, cudaBuf cudabuf, unsigned int size)
{
	COOLDAE(hipMemcpy(hostdata, cudabuf, size * sizeof(int), hipMemcpyDeviceToHost));
}

void transferAll(const int *a, const int *b, unsigned int size)
{
		cudaTransfer(deva, a, size);
		cudaTransfer(devb, b, size);
}

unsigned int cudaDo(int *c, unsigned int size)
{
		mulKernel << <1, size >> >(devc, deva, devb);

		// Check for any errors launching the kernel
		COOLDAEG(hipGetLastError());

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		COOLDAE(hipDeviceSynchronize());

		cudaGetBack(c, devc, size);

		return std::accumulate(c, c + size, 0);
}

unsigned int mulWithCudaHostSum(int *c, const int *a, const int *b, unsigned int size)
{
	//try {
		cudaSetup(size);
		transferAll(a, b, size);

		auto sum = cudaDo(c, size);

		// hipDeviceReset must be called before exiting in order for profiling and
		// tracing tools such as Nsight and Visual Profiler to show complete traces.
		COOLDAE(hipDeviceReset());

		return sum;

		/*
	} catch (std::exception e) {
		hipFree(devc);
		hipFree(devb);
		hipFree(deva);

		printf(e.what());
		throw e;
	}
	*/
}