#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <exception>

#define COOLDAE(x) { if (x != hipSuccess) throw std::exception(#x); }
#define COOLDAEG(x) { hipError_t status; status = x; if (status != hipSuccess) throw std::exception(hipGetErrorString(status)); }
#define COOLDAG(x, l) { if (x != hipSuccess) { fprintf(stderr, #x); goto l; }}

__global__ void mulKernel(int *productbuf, const int *arg1buf, const int *arg2buf)
{
	int i = threadIdx.x;
	productbuf[i] = arg1buf[i] * arg2buf[i];
}

typedef void* cudaBuf;

void cudaSetup(cudaBuf *a, cudaBuf *b, cudaBuf *c, unsigned int size)
{
	COOLDAE(hipSetDevice(0));
	COOLDAE(hipMalloc(a, size * sizeof(int)));
	COOLDAE(hipMalloc(b, size * sizeof(int)));
	COOLDAE(hipMalloc(c, size * sizeof(int)));
}

void cudaTeardown(cudaBuf a, cudaBuf b, cudaBuf c)
{
	hipFree(a);
	hipFree(b);
	hipFree(c);
}

void cudaTransfer(cudaBuf cudabuf, const int *hostdata, unsigned int size)
{
	COOLDAE(hipMemcpy(cudabuf, hostdata, size * sizeof(int), hipMemcpyHostToDevice));
}

void cudaGetBack(int *hostdata, cudaBuf cudabuf, unsigned int size)
{
	COOLDAE(hipMemcpy(hostdata, cudabuf, size * sizeof(int), hipMemcpyDeviceToHost));
}


void mulWithCudaHostSum(int *c, const int *a, const int *b, unsigned int size)
{
	int *deva = nullptr;
	int *devb = nullptr;
	int *devc = nullptr;

	try {

		cudaSetup((void**)&deva, (void**)&devb, (void**)&devc, size);
		cudaTransfer(deva, a, size);
		cudaTransfer(devb, b, size);

		mulKernel << <1, size >> >(devc, deva, devb);

		// Check for any errors launching the kernel
		COOLDAEG(hipGetLastError());

		// hipDeviceSynchronize waits for the kernel to finish, and returns
		// any errors encountered during the launch.
		COOLDAE(hipDeviceSynchronize());

		cudaGetBack(c, devc, size);

	} catch (std::exception e) {
		hipFree(devc);
		hipFree(devb);
		hipFree(deva);

		throw e;
	}
}

int main()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 10, 10, 10, 10 };
    int c[arraySize] = { 0 };

	mulWithCudaHostSum(c, a, b, arraySize);

    printf("{1,2,3,4,5} dot {10,10,10,10,10} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	COOLDAE(hipDeviceReset());

    return 0;
}